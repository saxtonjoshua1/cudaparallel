#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : monte_integrate.cu
 Author      : Joshua Saxton Kennesaw State University
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <hiprand/hiprand_kernel.h>

#define MAX_BLOCK_SIZE 1024
#define THREAD_ITERATIONS 64

__device__ float function(float x)
{
	return x*x;
}

void Get_args(int argc, char* argv[], float* b_input, int* threads_per_block_input, int* blocks_input)
{
   if (argc != 3) {
      fprintf(stderr, "usage: %s <b - integration limit> <blocks> <threads per block>\n",
            argv[0]);
      exit(0);
   }

   *b_input = strtod(argv[1], NULL);
   *blocks_input = strtol(argv[2], NULL, 10);
   *threads_per_block_input = strtol(argv[3], NULL, 10);
}

__global__ void integrate_kernel(float *estimate, hiprandState *states, float *b) {

		unsigned int thread = threadIdx.x + blockDim.x*blockIdx.x;

		int points_in = 0;
		float x, y;
		// Initialize CURAND

		hiprand_init(thread, 0, 0, &states[thread]);

		for(int i = 0; i < THREAD_ITERATIONS; i++) {
			x = hiprand_uniform(&states[thread]);
			y = hiprand_uniform(&states[thread]);
			// count if x & y is under the function
			if(y < function(x))
			{
				points_in++;
			}
		}
		//not sure how to compute a bounding box from 0 to b but maybe a simple square works?
		estimate[thread] =  (b * b * points_in) / (float) THREAD_ITERATIONS;

}

int main(int argc, char* argv[])
{
	int threads_per_block, blocks;
	float *b;
	float host[blocks * threads_per_block];
	float *device_result;
	hiprandState *devStates;

    Get_args(argc, argv, &b, &threads_per_block, &blocks);

    hipMalloc((void **) &device_result, blocks * threads_per_block * sizeof(float));
    hipMalloc((void **) &devStates, blocks*threads_per_block * sizeof(hiprandState));

    integrate_kernel<<<blocks, threads_per_block>>>(device_result, devStates,b);

    hipMemcpy(host, device_result, blocks * threads_per_block * sizeof(float),hipMemcpyDeviceToHost);

    // on the cpu compute a partial sum sequentially.
    float result=0.0;
    for(int i = 0; i < blocks * threads_per_block; i++)
		 result += host[i];
		 result /= (blocks * threads_per_block);
    printf("Freeing memory");
	hipFree(device_result);
	hipFree(devStates);
	printf("Returning result");
    return result;


}


